#include <cstdio>
#include <hip/hip_runtime.h>


/**
 * @brief display gpu information
 * @param devProp device
*/
void dispGPUInfo(const hipDeviceProp_t& devProp);
/**
 * @brief display gpu information
 * @param dev_id gpu id
 * @return GPU information
*/
hipDeviceProp_t getGPUInfo(const unsigned int& dev_id);

hipDeviceProp_t getGPUInfo(const unsigned int& dev_id)
{
	std::printf("----------------GPU----------------\r\n");
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, dev_id);
	return devProp;
}
void dispGPUInfo(const hipDeviceProp_t& devProp)
{
	std::printf("ʹGPU name: %s\r\n", devProp.name);
	std::printf("count of SMs: %d\r\n", devProp.multiProcessorCount);
	std::printf("capacity of shared memory per block: %f KB\r\n", devProp.sharedMemPerBlock / 1024.0);
	std::printf("max number of thread per block: %d\r\n", devProp.maxThreadsPerBlock);
	std::printf("max number of thread per SM: %d\r\n", devProp.maxThreadsPerMultiProcessor);
	std::printf("warp size: %d\r\n", devProp.warpSize);
	std::printf("max number of thread per SM per warp size: %d\r\n", 
		devProp.maxThreadsPerMultiProcessor / devProp.warpSize);
}